﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "wb.h"


int main(int argc, char* argv[]) {
    wbArg_t args;
    float* hostInput1 = nullptr;
    float* hostInput2 = nullptr;
    float* hostOutput = nullptr;
    float* deviceInput1;
    float* deviceInput2;
    float* deviceOutput;


    int inputLength;

    args = wbArg_read(argc, argv); /* чтение входных аргументов */

    // Импорт входных данных на хост
    wbTime_start(Generic, "Importing data to host");
    hostInput1 =(float*)wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 =(float*)wbImport(wbArg_getInputFile(args, 1), &inputLength);
    wbTime_stop(Generic, "Importing data to host");

    // Объявление и выделение памяти под выходные данные
    hostOutput = (float*)malloc(inputLength * sizeof(float));
    wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
    

    // Объявление и выделение памяти под входные и выходные данные  на устройства через thrust
    wbTime_start(GPU, "Doing GPU memory allocation");
   
    hipMalloc(&deviceInput1, inputLength * sizeof(float));
    hipMalloc(&deviceInput2, inputLength * sizeof(float));
    hipMalloc(&deviceOutput, inputLength * sizeof(float));

    thrust::device_ptr<float> d_in1(deviceInput1);
    thrust::device_ptr<float> d_in2(deviceInput2);
    thrust::device_ptr<float> d_out(deviceOutput);

    wbTime_stop(GPU, "Doing GPU memory allocation");

    // Копирование на устройство
    wbTime_start(Copy, "Copying data to the GPU");
    hipMemcpy(deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(deviceOutput, hostOutput, inputLength * sizeof(float), hipMemcpyHostToDevice);
    wbTime_stop(Copy, "Copying data to the GPU");

    // Выполнение операции сложения векторов
    wbTime_start(Compute, "Doing the computation on the GPU");
    thrust::transform(d_in1, d_in1 + inputLength * sizeof(float), d_in2, d_out, thrust::plus<float>());
    /////////////////////////////////////////////////////////

    // Копирование данных обратно на хост
    wbTime_start(Copy, "Copying data from the GPU");
    hipMemcpy(hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost);
    wbTime_stop(Copy, "Copying data from the GPU");

    wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);
    return 0;
}

